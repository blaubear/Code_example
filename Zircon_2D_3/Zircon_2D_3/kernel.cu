#include "hip/hip_runtime.h"
﻿//#include "Utilities.cuh"
#include "stdio.h"
#include "math.h"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/***********************************/
/* ITERATION FUNCTION - GPU */
/***********************************/
__global__ 
void Calculator_GPU(float* T_old, float* T_new, const int NX, const int NY, double D, double dx, double dt, double C_sat)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < NX-1 && i > 0 && j < NY-1 && j > 0)
        T_new[i + NY * j] = T_old[i + NY * j] + (dt/(dx*dx))* D * (T_old[i - 1 + NY * j] + T_old[i + 1 + NY * j] + T_old[i + (j - 1) * NY] + T_old[i + (j + 1) * NY] - 4.0 * T_old[i + NY * j]);
    if (i == 0)
        //T_new[i + NY * j] = T_new[i + 1 + NY * j];
        T_new[i + NY * j] = C_sat;
    if (i == (NX - 1) )
        //T_new[i + NY * j] = T_new[i - 1 + NY * j];
        T_new[i + NY * j] = C_sat;
    if (j == 0)
        //T_new[i + NY * j] = T_new[i + NY * (j + 1)];
        T_new[i + NY * j] = C_sat;
    if (j == (NY-1))
        //T_new[i + NY * j] = T_new[i + NY * (j - 1)];
        T_new[i + NY * j] = C_sat;
}


/***********************************/
/* ITERATION FUNCTION - GPU */
/***********************************/
__global__
void Calculator_GPU_X(float* T_old, float* T_new, const int NX, const int NY, double D, double dx, double dt, double C_sat)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    D = 0.1;
    if (i < NX - 1 && i > 0 && j < NY - 1 && j > 0)
        T_new[i + NY * j] = T_old[i + NY * j] + (dt / (dx * dx)) * D * (T_old[i - 1 + NY * j] + T_old[i + 1 + NY * j] + T_old[i + (j - 1) * NY] + T_old[i + (j + 1) * NY] - 4.0 * T_old[i + NY * j]);
    if (i == 0)
        //T_new[i + NY * j] = T_new[i + 1 + NY * j];
        T_new[i + NY * j] = C_sat;
    if (i == (NX - 1))
        T_new[i + NY * j] = T_new[i - 1 + NY * j];
    if (j == 0)
        //T_new[i + NY * j] = T_new[i + NY * (j + 1)];
        T_new[i + NY * j] = C_sat;
    if (j == (NY - 1))
        T_new[i + NY * j] = T_new[i + NY * (j - 1)];
}

/***********************************/
/* ITERATION FUNCTION - GPU */
/***********************************/
__global__
void Calculator_GPU_Y(float* T_old, float* T_new, const int NX, const int NY, float D, float dx, float dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    D = 0.1;
    if (i < NX - 1 && i > 0 && j < NY - 1 && j > 0)
        T_new[i + NY * j] = T_old[i + NY * j] + (dt / (dx * dx)) * D * (T_old[i - 1 + NY * j] + T_old[i + 1 + NY * j] + T_old[i + (j - 1) * NY] + T_old[i + (j + 1) * NY] - 4.0 * T_old[i + NY * j]);
    if (i == 0)
        //T_new[i + NY * j] = T_new[i + 1 + NY * j];
        T_new[i + NY * j] = 0;
    if (i == (NX - 1))
        T_new[i + NY * j] = T_new[i - 1 + NY * j];
    if (j == 0)
        //T_new[i + NY * j] = T_new[i + NY * (j + 1)];
        T_new[i + NY * j] = 0;
    if (j == (NY - 1))
        T_new[i + NY * j] = T_new[i + NY * (j - 1)];
}

/***********************************/
/*        Progonka - GPU      */
/***********************************/
__global__
void pronochka(float* T_old, float* T_new, const int NX, const int NY, float D, float dx, float dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    D = 0.1;
    if (i < NX - 1 && i > 0 && j < NY - 1 && j > 0)
        T_new[i + NY * j] = T_old[i + NY * j] + (dt / (dx * dx)) * D * (T_old[i - 1 + NY * j] + T_old[i + 1 + NY * j] + T_old[i + (j - 1) * NY] + T_old[i + (j + 1) * NY] - 4.0 * T_old[i + NY * j]);
    if (i == 0)
        //T_new[i + NY * j] = T_new[i + 1 + NY * j];
        T_new[i + NY * j] = 0;
    if (i == (NX - 1))
        T_new[i + NY * j] = T_new[i - 1 + NY * j];
    if (j == 0)
        //T_new[i + NY * j] = T_new[i + NY * (j + 1)];
        T_new[i + NY * j] = 0;
    if (j == (NY - 1))
        T_new[i + NY * j] = T_new[i + NY * (j - 1)];
}


/******************************/
/* TEMPERATURE INITIALIZATION */
/******************************/

void Initialize(float* h_T, const int NX, const int NY)
{
    for (int i = 0; i < NX; i++) {
        for (int j = 0; j < NY; j++) {
            //h_T[i * NY +j ] = i/128.0;
            h_T[i * NY + j] = 300;
        }
    }
}

/******************************/
/*      Write in file 1       */
/******************************/
void WriteInFile_1(float* h_T_GPU_result, int NX, int NY) {
    // --- Write in file
    FILE* pointer = fopen("1.txt", "w");
    for (int i = 0; i < NX; i++) {
        for (int j = 0; j < NY; j++) {
            fprintf(pointer, "%f ", h_T_GPU_result[j * NX + i]);
        }
        putc('\n', pointer);
    }
    fclose(pointer);
}

/******************************/
/*      Write in file 2       */
/******************************/
void WriteInFile_2(float* h_T_GPU_result, int NX, int NY) {
    // --- Write in file
    FILE* pointer = fopen("2.txt", "w");
    for (int i = 0; i < NX; i++) {
        for (int j = 0; j < NY; j++) {
            fprintf(pointer, "%f ", h_T_GPU_result[j * NX + i]);
        }
        putc('\n', pointer);
    }
    fclose(pointer);
}

/********/
/* MAIN */
/********/
int main(){
    const int NX = 512;         // --- Number of discretization points along the x axis
    const int NY = NX;         // --- Number of discretization points along the y axis
    int Nt = 200000, MAX_ITER = 1;
    double Lx, Ly, D = 0.1;
    Lx = 0.1;
    Ly = Lx;
    double dx = Lx / NX;
    double T_start = 950 + 273; // начальная температура в кельвинах;
    double T_end = 750 + 273;  //конечная температура в кельвинах;
    double time_coef = (T_start - T_end);
    double t_end = 10000 * 365 * 24 * 60 * 60; // окончание по времени;
    //double t_end = 1;
    double X_H20 = 2;
    double dt = 1 / (double)Nt;
    double T, M, D_nd;
    double time = 0;
    double C_sat = 100, C_cryst = 490000;
    int Counter = 0;
    hipEvent_t start, stop;
    float elapsedTime;

    float* GPU_D ;
    hipMalloc((void**)&GPU_D, sizeof(float));
    hipEventCreate(&start);
    hipEventRecord(start, 0);

    // --- GPU temperature distribution
    float* h_T = (float*)calloc(NX * NY, sizeof(float));
    float* h_T_old = (float*)calloc(NX * NY, sizeof(float));
    Initialize(h_T, NX, NY);
    Initialize(h_T_old, NX, NY);
    float* h_T_GPU_result = (float*)malloc(NX * NY * sizeof(float));

    WriteInFile_1(h_T, NX, NY);

    // --- GPU temperature distribution
    float* d_T;     hipMalloc((void**)&d_T, NX * NY * sizeof(float));
    float* d_T_old; hipMalloc((void**)&d_T_old, NX * NY * sizeof(float));

    hipMemcpy(d_T, h_T, NX * NY * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_T_old, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToDevice);
    
    dim3 threadsPerBlock(32, 32);
    int nbx = (NX / threadsPerBlock.x) + (((NX % threadsPerBlock.x) == 0) ? 0 : 1);
    int nby = (NY / threadsPerBlock.y) + (((NY % threadsPerBlock.y) == 0) ? 0 : 1);
    dim3 numBlocks(nbx, nby);

    // --- Iterations on the device
    for (int i = 0; i < Nt; i++) {
        Counter = Counter + 1;
        time = time + dt;
        T = T_start - time_coef * time;
        M = 4.8 * pow(10, -6) * pow(T, 2) - 8.4 * pow(10, -3) * T + 4.84;
        C_sat = C_cryst / (exp(10108 / T + 1.16 * (M - 1) - 1.48));

        //a = fzero(@(x_nd)pi ^ (1 / 2) * x_nd * exp(x_nd ^ 2) * erfc(x_nd) - (C_bound - C_sat) / (C_cryst - C_sat), 0);% вычисление промежуточной величины для аналитического решения;
        D = (exp(-(11.4 * X_H20 + 3.13) / (0.84 * X_H20 + 1) - ((21.4 * X_H20 + 47) / (1.06 * X_H20 + 1)) * (1000) / T));
        D_nd = D * t_end / pow(Lx, 2);
        Calculator_GPU << <numBlocks, threadsPerBlock >> > (d_T, d_T_old, NX, NY, D_nd, dx, dt, C_sat);   // --- Update d_T_old     starting from data stored in d_T

        //hipMemcpy(d_T_old, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToDevice);
        d_T_old = d_T;
        if (fmod(Counter, Nt/10) == 0) {
        printf("%f \n", time);
        }
    }
    gpuErrchk(hipPeekAtLastError());
    // --- Copy result from device to host
    hipMemcpy(h_T_GPU_result, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToHost);

    // --- Write in file 2
    WriteInFile_2(h_T_GPU_result, NX, NY);

    // --- Release device memory
    hipFree(d_T);
    hipFree(d_T_old);

    free(h_T);
    free(h_T_GPU_result);

    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed time : %f ms\n", elapsedTime);


    return 0;
}
